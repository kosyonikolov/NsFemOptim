#include "hip/hip_runtime.h"
#include <NavierStokes/chorinCuda.h>

#include <cassert>
#include <format>
#include <stdexcept>

#include <cu/blas.h>
#include <cu/csrF.h>
#include <cu/solvers/solverFactory.h>
#include <cu/sparse.h>
#include <cu/spmv.h>
#include <cu/spmm.h>

#include <linalg/io.h>

#include <utils/stopwatch.h>

#include <NavierStokes/buildContext.h>

struct DirichletVelocity
{
    cu::Sparse & lib;

    cu::vec<float> & velocityXy; // X, then Y
    cu::vec<int> ids;            // size = n
    cu::vec<float> values;       // size = n

    hipsparseDnVecDescr_t denseVec;  // velocityXy
    hipsparseSpVecDescr_t sparseVec; // ids, values

    DirichletVelocity(cu::Sparse & sparseHandle,
                      cu::vec<float> & velocityXy,
                      const std::vector<fem::DirichletNode> & x,
                      const std::vector<fem::DirichletNode> & y)
        : lib(sparseHandle), velocityXy(velocityXy)
    {
        const int numNodes = velocityXy.size();
        assert(numNodes % 2 == 0);
        const int numNodesChannel = numNodes / 2;

        const int nnz = x.size() + y.size();
        std::vector<int> cpuIds(nnz);
        std::vector<float> cpuVals(nnz);
        int i = 0;
        for (int j = 0; j < x.size(); j++, i++)
        {
            cpuIds[i] = x[j].id;
            cpuVals[i] = x[j].value;
        }
        for (int j = 0; j < y.size(); j++, i++)
        {
            cpuIds[i] = y[j].id + numNodesChannel;
            cpuVals[i] = y[j].value;
        }

        ids.overwriteUpload(cpuIds);
        values.overwriteUpload(cpuVals);

        auto rc = hipsparseCreateSpVec(&sparseVec, numNodes, nnz, ids.get(), values.get(),
                                      hipsparseIndexType_t::HIPSPARSE_INDEX_32I,
                                      hipsparseIndexBase_t::HIPSPARSE_INDEX_BASE_ZERO,
                                      hipDataType::HIP_R_32F);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to create cusparse sparse vector: {}", hipsparseGetErrorName(rc)));
        }

        denseVec = velocityXy.getCuSparseDescriptor();
    }

    void impose()
    {
        auto rc = hipsparseScatter(lib.handle(), sparseVec, denseVec);
    }
};

struct PressureSolver
{
    cu::Sparse & lib;

    cu::AbstractSolver & solver;

    int numAll;
    int numInternal;

    // Input/output buffer
    // Before pressure is calculated, this is tentativeVelDiv
    // After it is calculated, this is the pressure
    cu::vec<float> dense;
    cu::vec<int> internalIds;

    hipsparseSpVecDescr_t sparseInput;  // values = rhs
    hipsparseSpVecDescr_t sparseOutput; // values = internalPressure

    PressureSolver(cu::AbstractSolver & solver, cu::Sparse & lib,
                   const int numPressureNodes,
                   const std::vector<int> & internalPressureIds)
        : lib(lib), solver(solver),
          dense(numPressureNodes),
          internalIds(internalPressureIds)
    {
        numAll = numPressureNodes;
        numInternal = internalPressureIds.size();
        assert(numInternal > 0 && numInternal <= numAll);

        auto & rhs = solver.getRhs();
        auto & internalPressure = solver.getSol();

        auto rc = hipsparseCreateSpVec(&sparseInput, numAll, numInternal,
                                      internalIds.get(),
                                      rhs.get(),
                                      hipsparseIndexType_t::HIPSPARSE_INDEX_32I,
                                      hipsparseIndexBase_t::HIPSPARSE_INDEX_BASE_ZERO,
                                      hipDataType::HIP_R_32F);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseCreateSpVec failed: {}", hipsparseGetErrorName(rc)));
        }

        rc = hipsparseCreateSpVec(&sparseOutput, numAll, numInternal,
                                 internalIds.get(),
                                 internalPressure.get(),
                                 hipsparseIndexType_t::HIPSPARSE_INDEX_32I,
                                 hipsparseIndexBase_t::HIPSPARSE_INDEX_BASE_ZERO,
                                 hipDataType::HIP_R_32F);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseCreateSpVec failed: {}", hipsparseGetErrorName(rc)));
        }
    }

    void update()
    {
        auto rc = hipsparseGather(lib.handle(), dense.getCuSparseDescriptor(), sparseInput);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseGather failed: {}", hipsparseGetErrorName(rc)));
        }

        // solver.rhs is now updated
        solver.solve();
        // solver.sol is now updated

        // Output pressure
        dense.memsetZero();
        rc = hipsparseScatter(lib.handle(), sparseOutput, dense.getCuSparseDescriptor());
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseScatter failed: {}", hipsparseGetErrorName(rc)));
        }
    }
};

Solution solveNsChorinCuda(const mesh::ConcreteMesh & velocityMesh, const mesh::ConcreteMesh & pressureMesh,
                           const DfgConditions & cond, const float timeStep0, const float maxT,
                           const ChorinCudaConfig & cfg)
{
    cu::Blas blas;
    cu::Sparse sparse;

    float plusOne = 1.0f;

    auto ctx = buildChorinContext(velocityMesh, pressureMesh, cond);

    // Create CUDA matrices
    cu::csrF velocityMass(ctx.velocityMass);
    cu::csrF velocityStiffnessPlusConvection(ctx.velocityStiffness);
    cu::csrF pressureStiffnessInternal(ctx.pressureStiffnessInternal);
    cu::csrF velocityPressureDiv(ctx.velocityPressureDiv);
    cu::csrF pressureVelocityDiv(ctx.pressureVelocityDiv);
    cu::csrF fastConvectionIntegration(ctx.fastConvectionIntegration);

    // Copy original stiffness matrix values
    // On each iteration we will do A = viscosity * M1 + C and store the result in velocityStiffnessPlusConvection
    cu::vec<float> origVelocityM1Vals(velocityStiffnessPlusConvection.values);
    float viscosity = cond.viscosity;
    auto blasRc = hipblasSscal(blas.handle, origVelocityM1Vals.size(), &viscosity, origVelocityM1Vals.get(), 1);
    if (blasRc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
    {
        throw std::runtime_error(std::format("Failed to scale M1: {}", cublasGetStatusName(blasRc)));
    }

    cu::spmv fcSpmv(sparse.handle(), fastConvectionIntegration);
    cu::spmm aSpmm(sparse.handle(), velocityStiffnessPlusConvection, 2);

    const int numVelocityNodes = ctx.numVelocityNodes;
    const int numPressureNodes = ctx.numPressureNodes;

    cu::vec<float> velocityXy(2 * numVelocityNodes); // X, then Y
    velocityXy.memsetZero();

    DirichletVelocity dirichletVelocity(sparse, velocityXy, ctx.dirichletVx, ctx.dirichletVy);
    dirichletVelocity.impose();
    // Create overlapping views of the X and Y velocities
    cu::vec<float> velocityX(velocityXy.get(), numVelocityNodes);
    cu::vec<float> velocityY(velocityXy.get() + numVelocityNodes, numVelocityNodes);

    // Acceleration
    // cu::vec<float> accel(2 * numVelocityNodes);
    const auto & vSolverCfg = cfg.velocitySolver;
    auto velocitySolver = cu::createSolver(vSolverCfg.method, 2,
                                           ctx.velocityMass, vSolverCfg.maxIterations,
                                           vSolverCfg.targetMse, vSolverCfg.mseCheckInterval);
    auto & accel = velocitySolver->getSol();

    // Pressure
    cu::spmv vpdSpmv(sparse.handle(), velocityPressureDiv);
    const auto & pSolverCfg = cfg.pressureSolver;
    auto pressureSolverCore = cu::createSolver(pSolverCfg.method, 1,
                                               ctx.pressureStiffnessInternal, pSolverCfg.maxIterations,
                                               pSolverCfg.targetMse, pSolverCfg.mseCheckInterval);
    PressureSolver pressureSolver(*pressureSolverCore, sparse, numPressureNodes, ctx.internalPressureNodes);

    cu::spmv pvdSpmv(sparse.handle(), pressureVelocityDiv);
    auto & nablaPXy = pvdSpmv.b;
    assert(nablaPXy.size() == 2 * numVelocityNodes);
    // Create vectors for the X and Y components of nabla
    cu::vec<float> nablaPX(nablaPXy.get(), numVelocityNodes);
    cu::vec<float> nablaPY(nablaPXy.get() + numVelocityNodes, numVelocityNodes);

    const int numTimeSteps = std::ceil(maxT / timeStep0);
    const float tau = maxT / numTimeSteps;
    const float invTau = -1.0f / tau;
    Solution result;
    result.steps.resize(numTimeSteps + 1);

    // ======= Debug dumps =======
    const std::string dumpDir = "dumps_cuda";
    const bool dbgDumps = false;
    std::vector<float> dbgVelocityXy(velocityXy.size());
    // std::vector<float> dbgPressureRhs(pressureSolver.rhs.size());
    std::vector<float> dbgPressureRhs(pressureSolver.solver.getRhs().size());
    std::vector<float> dbgInternalP(ctx.internalPressureNodes.size());
    std::vector<float> dbgFullP(numPressureNodes);

    for (int iT = 0; iT <= numTimeSteps; iT++)
    {
        u::Stopwatch bigSw;
        u::Stopwatch sw;

        // Update convection
        auto & currConvection = fcSpmv.b;
        fcSpmv.compute(velocityXy, currConvection);
        // Calculate A = viscosity * M1 + convection
        auto & aValues = velocityStiffnessPlusConvection.values;
        blasRc = hipblasSgeam(blas.handle, hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_N,
                             aValues.size(), 1,
                             &plusOne, origVelocityM1Vals.get(), aValues.size(),
                             &plusOne, currConvection.get(), aValues.size(),
                             aValues.get(), aValues.size());
        if (blasRc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipblasSgeam failed: {}", cublasGetStatusName(blasRc)));
        }
        const auto tConvection = sw.millis(true);

        // =========================================================================================
        // Find tentative velocity in two steps:
        // 1) Compute accelRhsC = A * velocityC
        // 2) Solve M0 * accelC = accelRhsC
        // Solve for X and Y simultaneously

        auto & accelRhs = velocitySolver->getRhs();
        aSpmm.compute(velocityXy, accelRhs);
        accel.memsetZero();
        velocitySolver->solve();
        const float mseTentX = velocitySolver->getLastMse(0);
        const float mseTentY = velocitySolver->getLastMse(1);
        const int tentIters = velocitySolver->getLastIterations();
        if (!std::isfinite(mseTentX) || !std::isfinite(mseTentY))
        {
            throw std::runtime_error("Tentative acceleration is bad");
        }
        // v* = v - tau * accel
        cu::saxpy(blas, 2 * numVelocityNodes, accel.get(), velocityXy.get(), -tau);

        // Reimpose BCs
        dirichletVelocity.impose();
        if (dbgDumps)
        {
            velocityXy.download(dbgVelocityXy);
            linalg::write(std::format("{}/{}_tentativeVxy.bin", dumpDir, iT), dbgVelocityXy);
        }
        const auto tTentative = sw.millis(true);
        // =========================================================================================

        // =========================================================================================
        // Find the pressure:
        // 1) Compute the RHS of the system:
        //      tentativeVelDiv = velocityPressureDiv * tentativeVelocityXy;
        //      pressureRhs = gather(tenativeVelDiv, internalPressureNodes)
        // 2) Find the internal pressure: pressureInt = pressureStiffnessSolver.solve(pressureRhs);
        // 3) Scatter the internal pressure

        // delta(p) = nabla . u_* / tau
        // Calculate the divergence of the tentative velocity
        vpdSpmv.compute(velocityXy, pressureSolver.dense);
        cu::scale(blas, pressureSolver.dense.size(), pressureSolver.dense.get(), invTau);

        pressureSolver.update();
        const float msePressure = pressureSolver.solver.getLastMse();
        const int pressureIters = pressureSolver.solver.getLastIterations();
        if (dbgDumps)
        {
            // pressureSolver.rhs.download(dbgPressureRhs);
            pressureSolver.solver.getRhs().download(dbgPressureRhs);
            linalg::write(std::format("{}/{}_pressureRhs.bin", dumpDir, iT), dbgPressureRhs);
        }

        auto & pressure = pressureSolver.dense;
        assert(pressure.size() == numPressureNodes);

        if (dbgDumps)
        {
            // pressureSolver.internalPressure.download(dbgInternalP);
            pressureSolver.solver.getSol().download(dbgInternalP);
            pressure.download(dbgFullP);
            linalg::write(std::format("{}/{}_internalP.bin", dumpDir, iT), dbgInternalP);
            linalg::write(std::format("{}/{}_fullP.bin", dumpDir, iT), dbgFullP);
        }

        const auto tPressure = sw.millis(true);

        // Copy to output
        auto & outP = result.steps[iT].pressure;
        outP.resize(numPressureNodes);
        pressure.download(outP);

        const auto tPressureDownload = sw.millis();
        // =========================================================================================

        // =========================================================================================
        // Find the final velocity by updating the tentative
        // (u_{i+1} - u_*) / tau = -nabla(p) <=>
        // <=> a = nabla(p) <=>
        // <=> (a, v) = (nabla(p), v)
        // Then update: u_{i+1} = u_* + tau * a
        // Calculate X and Y channels simultaneously

        // nablaPXy = pressureVelocityDiv * pressure;
        pvdSpmv.compute(pressure, nablaPXy);

        nablaPXy.copyTo(accelRhs); // TODO Can we compute in accelRhs directly?
        accel.memsetZero();
        velocitySolver->solve();
        const float mseFinalX = velocitySolver->getLastMse(0);
        const float mseFinalY = velocitySolver->getLastMse(1);
        const int finalIters = velocitySolver->getLastIterations();
        if (!std::isfinite(mseFinalX) || !std::isfinite(mseFinalY))
        {
            throw std::runtime_error("Final acceleration is bad");
        }

        cu::saxpy(blas, 2 * numVelocityNodes, accel.get(), velocityXy.get(), -tau);

        dirichletVelocity.impose();

        const float tFinal = sw.millis(true);

        // Copy to output
        auto & outVelocity = result.steps[iT].velocity;
        outVelocity.resize(velocityXy.size());
        velocityXy.download(outVelocity);

        const float tFinalDownload = sw.millis();
        const float tIter = bigSw.millis();
        // =========================================================================================

        std::cout << std::format("{} / {}: {} ms\n", iT, numTimeSteps, tIter);
        std::cout << std::format("\tconvection = {}, tentative = {}, pressure = {}, pressureDownload = {}, final = {}, finalDownload = {}\n",
                                 tConvection, tTentative, tPressure, tPressureDownload, tFinal, tFinalDownload);
        std::cout << std::format("\tMSEs: tent (X / Y / iters) = {} / {} / {}, pressure = {} / {}, final (X / Y / iters) = {} / {} / {}\n",
                                 mseTentX, mseTentY, tentIters,
                                 msePressure, pressureIters, 
                                 mseFinalX, mseFinalY, finalIters);
    }

    return result;
}