#include <cu/vec.h>

#include <cu/datatypes.h>

namespace cu
{
    template<typename F>
    hipsparseDnVecDescr_t vec<F>::getCuSparseDescriptor()
    {
        if (cuSparseDescriptor != 0)
        {
            return cuSparseDescriptor;
        }

        const auto dtype = getCudaDataType<F>();
        auto rc = hipsparseCreateDnVec(&cuSparseDescriptor, size(), get(), dtype);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to create vector descriptor: {}", hipsparseGetErrorName(rc)));
        }

        return cuSparseDescriptor;
    }

    template<typename F>
    hipsparseDnMatDescr_t vec<F>::getCuSparseMatDescriptor(const int numCh)
    {
        const auto n = size();
        if (numCh < 1 || n % numCh != 0)
        {
            throw std::invalid_argument(std::format("Bad number of channels [{}] for vector of size {}", numCh, n));
        }

        if (cuSparseMatDescriptor != 0)
        {
            return cuSparseMatDescriptor;
        }

        const auto dtype = getCudaDataType<F>();
        const size_t rows = n / numCh;
        const size_t cols = numCh;
        auto rc = hipsparseCreateDnMat(&cuSparseMatDescriptor, 
                                      rows, cols, rows, 
                                      get(), dtype, hipsparseOrder_t::HIPSPARSE_ORDER_COL);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to create dense mat descriptor: {}", hipsparseGetErrorName(rc)));
        }

        return cuSparseMatDescriptor;
    }

    template hipsparseDnVecDescr_t vec<float>::getCuSparseDescriptor();
    template hipsparseDnVecDescr_t vec<double>::getCuSparseDescriptor();

    template hipsparseDnMatDescr_t vec<float>::getCuSparseMatDescriptor(const int numCh);
    template hipsparseDnMatDescr_t vec<double>::getCuSparseMatDescriptor(const int numCh);
}