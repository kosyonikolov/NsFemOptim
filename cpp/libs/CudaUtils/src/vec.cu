#include <cu/vec.h>

#include <cu/datatypes.h>

namespace cu
{
    template<typename F>
    hipsparseDnVecDescr_t vec<F>::getCuSparseDescriptor()
    {
        if (cuSparseDescriptor != 0)
        {
            return cuSparseDescriptor;
        }

        const auto dtype = getCudaDataType<F>();
        auto rc = hipsparseCreateDnVec(&cuSparseDescriptor, size(), get(), dtype);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to create vector descriptor: {}", hipsparseGetErrorName(rc)));
        }

        return cuSparseDescriptor;
    }

    template hipsparseDnVecDescr_t vec<float>::getCuSparseDescriptor();
    template hipsparseDnVecDescr_t vec<double>::getCuSparseDescriptor();
}