#include <cu/hipsparse.h>

#include <stdexcept>
#include <format>
#include <iostream>
#include <cassert>

namespace cu
{
    Sparse::Sparse()
    {
        auto rc = hipsparseCreate(&theHandle);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to create cusparse: {}", hipsparseGetErrorName(rc)));
        }
    }

    Sparse::~Sparse()
    {
        auto rc = hipsparseDestroy(theHandle);
        assert(rc == HIPSPARSE_STATUS_SUCCESS);
        if (rc != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cerr << "Failed to destroy cusparse handle: " << hipsparseGetErrorName(rc) << "\n";
        }
    }
}