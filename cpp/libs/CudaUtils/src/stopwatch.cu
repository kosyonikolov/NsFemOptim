#include <cu/stopwatch.h>

#include <format>
#include <stdexcept>

namespace cu
{
    Stopwatch::Stopwatch(hipStream_t stream)
        : stream(stream)
    {
        auto rc = hipEventCreate(&start);
        if (rc != hipError_t::hipSuccess)
        {
            throw std::runtime_error(std::format("hipEventCreate failed: {}", hipGetErrorName(rc)));
        }

        rc = hipEventCreate(&stop);
        if (rc != hipError_t::hipSuccess)
        {
            throw std::runtime_error(std::format("hipEventCreate failed: {}", hipGetErrorName(rc)));
        }

        rc = hipEventRecord(start, stream);
        if (rc != hipError_t::hipSuccess)
        {
            throw std::runtime_error(std::format("hipEventRecord failed: {}", hipGetErrorName(rc)));
        }
    }

    void Stopwatch::reset()
    {
        auto rc = hipEventRecord(start, stream);
        if (rc != hipError_t::hipSuccess)
        {
            throw std::runtime_error(std::format("hipEventRecord failed: {}", hipGetErrorName(rc)));
        }
    }

    float Stopwatch::millis(const bool reset)
    {
        auto rc = hipEventRecord(stop, stream);
        if (rc != hipError_t::hipSuccess)
        {
            throw std::runtime_error(std::format("hipEventRecord failed: {}", hipGetErrorName(rc)));
        }

        rc = hipEventSynchronize(stop);
        if (rc != hipError_t::hipSuccess)
        {
            throw std::runtime_error(std::format("hipEventSynchronize failed: {}", hipGetErrorName(rc)));
        }

        float ms = -1;
        rc = hipEventElapsedTime(&ms, start, stop);
        if (rc != hipError_t::hipSuccess)
        {
            throw std::runtime_error(std::format("hipEventElapsedTime failed: {}", hipGetErrorName(rc)));
        }

        if (reset)
        {
            rc = hipEventRecord(start, stream);
            if (rc != hipError_t::hipSuccess)
            {
                throw std::runtime_error(std::format("hipEventRecord failed: {}", hipGetErrorName(rc)));
            }
        }

        return ms;
    }
} // namespace cu