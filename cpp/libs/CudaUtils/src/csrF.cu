#include <cu/csrF.h>

#include <cu/hipsparse.h>

namespace cu
{
    csrF::csrF(const linalg::CsrMatrix<float> & cpuMat)
        : rows(cpuMat.rows), cols(cpuMat.cols),
          values(cpuMat.values), column(cpuMat.column), rowStart(cpuMat.rowStart)
    {
    }

    csrF::~csrF()
    {
        if (matDesc)
        {
            auto rc = hipsparseDestroySpMat(matDesc);
            assert(rc == hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS);
        }
    }

    hipsparseSpMatDescr_t csrF::getCuSparseDescriptor()
    {
        if (matDesc != 0)
        {
            return matDesc;
        }

        auto rc = hipsparseCreateCsr(&matDesc, rows, cols,
                                    values.size(), rowStart.get(),
                                    column.get(), values.get(),
                                    hipsparseIndexType_t::HIPSPARSE_INDEX_32I,
                                    hipsparseIndexType_t::HIPSPARSE_INDEX_32I,
                                    hipsparseIndexBase_t::HIPSPARSE_INDEX_BASE_ZERO,
                                    hipDataType::HIP_R_32F);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to create CSR: {}", hipsparseGetErrorName(rc)));
        }

        return matDesc;
    }
} // namespace cu