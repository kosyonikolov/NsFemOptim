#include <cu/csrF.h>

#include <cu/hipsparse.h>

namespace cu
{
    csrF::csrF(const linalg::CsrMatrix<float> & cpuMat)
        : rows(cpuMat.rows), cols(cpuMat.cols),
          values(cpuMat.values), column(cpuMat.column), rowStart(cpuMat.rowStart),
          x(cpuMat.cols), b(cpuMat.rows)
    {
        handle = getCuSparseHandle(); // TODO Support custom handles

        // Create cusparse descriptors
        auto rc = hipsparseCreateDnVec(&xDesc, x.size(), x.get(), hipDataType::HIP_R_32F);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to create x vector descriptor: {}", hipsparseGetErrorName(rc)));
        }

        rc = hipsparseCreateDnVec(&bDesc, b.size(), b.get(), hipDataType::HIP_R_32F);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to create b vector descriptor: {}", hipsparseGetErrorName(rc)));
        }

        rc = hipsparseCreateCsr(&matDesc, rows, cols,
                               values.size(), rowStart.get(),
                               column.get(), values.get(),
                               hipsparseIndexType_t::HIPSPARSE_INDEX_32I,
                               hipsparseIndexType_t::HIPSPARSE_INDEX_32I,
                               hipsparseIndexBase_t::HIPSPARSE_INDEX_BASE_ZERO,
                               hipDataType::HIP_R_32F);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to create CSR: {}", hipsparseGetErrorName(rc)));
        }

        size_t spmvBufferSize = 0;
        rc = hipsparseSpMV_bufferSize(handle, op,
                                     &alpha, matDesc, xDesc,
                                     &beta, bDesc,
                                     hipDataType::HIP_R_32F, hipsparseSpMVAlg_t::HIPSPARSE_SPMV_ALG_DEFAULT,
                                     &spmvBufferSize);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseSpMV_bufferSize failed: {}", hipsparseGetErrorName(rc)));
        }
        std::cout << "Workspace buffer size: " << spmvBufferSize << "\n";

        workspace = cu::vec<char>(spmvBufferSize);

        rc = hipsparseSpMV_preprocess(handle, op,
                                     &alpha, matDesc, xDesc, &beta, bDesc,
                                     hipDataType::HIP_R_32F, hipsparseSpMVAlg_t::HIPSPARSE_SPMV_ALG_DEFAULT,
                                     workspace.get());
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseSpMV_preprocess failed: {}", hipsparseGetErrorName(rc)));
        }
    }

    csrF::~csrF()
    {
        // TODO
    }

    void csrF::spmv()
    {
        return spmv(xDesc, bDesc);
    }

    void csrF::spmv(hipsparseDnVecDescr_t otherX, hipsparseDnVecDescr_t otherB)
    {
        auto rc = hipsparseSpMV(handle, op, &alpha,
                               matDesc, otherX, &beta, otherB,
                               hipDataType::HIP_R_32F, hipsparseSpMVAlg_t::HIPSPARSE_SPMV_ALG_DEFAULT,
                               workspace.get());
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseSpMV failed: {}", hipsparseGetErrorName(rc)));
        }
    }
} // namespace cu