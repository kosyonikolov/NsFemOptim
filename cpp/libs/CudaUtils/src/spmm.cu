#include <cu/spmm.h>

namespace cu
{
    spmm::spmm(hipsparseHandle_t handle, cu::csrF & m,
               const int numCh)
        : handle(handle), mat(m), numCh(numCh),
          x(m.cols * numCh), b(m.rows * numCh)
    {
        if (numCh < 1)
        {
            throw std::invalid_argument("SPMM channels must be at least 1");
        }

        matDesc = mat.getCuSparseDescriptor();

        auto xDesc = x.getCuSparseMatDescriptor(numCh);
        auto bDesc = b.getCuSparseMatDescriptor(numCh);

        size_t spmmBufferSize = 0;
        auto rc = hipsparseSpMM_bufferSize(handle, op, op,
                                          &alpha, matDesc, xDesc,
                                          &beta, bDesc,
                                          hipDataType::HIP_R_32F, hipsparseSpMMAlg_t::HIPSPARSE_SPMM_ALG_DEFAULT,
                                          &spmmBufferSize);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseSpMM_bufferSize failed: {}", hipsparseGetErrorName(rc)));
        }
        std::cout << "Workspace buffer size: " << spmmBufferSize << "\n";

        workspace = cu::vec<char>(spmmBufferSize);

        rc = hipsparseSpMM_preprocess(handle, op, op,
                                     &alpha, matDesc, xDesc, &beta, bDesc,
                                     hipDataType::HIP_R_32F, hipsparseSpMMAlg_t::HIPSPARSE_SPMM_ALG_DEFAULT,
                                     workspace.get());
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseSpMV_preprocess failed: {}", hipsparseGetErrorName(rc)));
        }
    }

    spmm::~spmm()
    {
    }

    void spmm::compute()
    {
        return compute(x.getCuSparseMatDescriptor(numCh), b.getCuSparseMatDescriptor(numCh));
    }

    void spmm::compute(hipsparseDnMatDescr_t otherX, hipsparseDnMatDescr_t otherB)
    {
        auto rc = hipsparseSpMM(handle, op, op,
                               &alpha, matDesc, otherX, &beta, otherB,
                               hipDataType::HIP_R_32F, hipsparseSpMMAlg_t::HIPSPARSE_SPMM_ALG_DEFAULT,
                               workspace.get());
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseSpMM failed: {}", hipsparseGetErrorName(rc)));
        }
    }

    void spmm::compute(cu::vec<float> & otherX, cu::vec<float> & otherB)
    {
        auto xd = otherX.getCuSparseMatDescriptor(numCh);
        auto bd = otherB.getCuSparseMatDescriptor(numCh);
        compute(xd, bd);
    }

} // namespace cu