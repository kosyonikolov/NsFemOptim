#include <format>
#include <stdexcept>
#include <cassert>
#include <iostream>

#include <cu/blas.h>

namespace cu
{
    Blas::Blas()
    {
        auto rc = hipblasCreate(&handle);
        if (rc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to create cublas handle: {}", cublasGetStatusName(rc)));
        }
    }

    Blas::~Blas()
    {
        if (handle)
        {
            auto rc = hipblasDestroy(handle);
            if (rc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
            {
                std::cerr << "Failed to destroy cublas handle: " << cublasGetStatusName(rc) << "\n";
            }
        }
    }

    void Blas::setStream(hipStream_t stream)
    {
        assert(handle);
        auto rc = hipblasSetStream(handle, stream);
        if (rc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to set cublas stream: {}", cublasGetStatusName(rc)));
        }
    }

    void Blas::setPointerMode(hipblasPointerMode_t pointerMode)
    {
        assert(handle);
        auto rc = hipblasSetPointerMode(handle, pointerMode);
        if (rc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to set cublas pointer mode: {}", cublasGetStatusName(rc)));
        }
    }
} // namespace cu