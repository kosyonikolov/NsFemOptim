#include <format>
#include <stdexcept>
#include <cassert>
#include <iostream>

#include <cu/blas.h>

namespace cu
{
    Blas::Blas()
    {
        auto rc = hipblasCreate(&handle);
        if (rc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to create cublas handle: {}", cublasGetStatusName(rc)));
        }
    }

    Blas::~Blas()
    {
        if (handle)
        {
            auto rc = hipblasDestroy(handle);
            if (rc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
            {
                std::cerr << "Failed to destroy cublas handle: " << cublasGetStatusName(rc) << "\n";
            }
        }
    }

    void Blas::setStream(hipStream_t stream)
    {
        assert(handle);
        auto rc = hipblasSetStream(handle, stream);
        if (rc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to set cublas stream: {}", cublasGetStatusName(rc)));
        }
    }

    void Blas::setPointerMode(hipblasPointerMode_t pointerMode)
    {
        assert(handle);
        auto rc = hipblasSetPointerMode(handle, pointerMode);
        if (rc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("Failed to set cublas pointer mode: {}", cublasGetStatusName(rc)));
        }
    }

    void saxpy(Blas & blas, const int n, float * src, float * dst, float alpha)
    {
        auto rc = hipblasSaxpy(blas.handle, n, &alpha, src, 1, dst, 1);
        if (rc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipblasSaxpy failed: {}", cublasGetStatusName(rc)));
        }
    }

    void scale(Blas & blas, const int n, float * dst, float alpha)
    {
        auto rc = hipblasSscal(blas.handle, n, &alpha, dst, 1);
        if (rc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipblasSscal failed: {}", cublasGetStatusName(rc)));
        }
    }
} // namespace cu