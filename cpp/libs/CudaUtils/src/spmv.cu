#include <cu/spmv.h>

namespace cu
{
    spmv::spmv(hipsparseHandle_t handle, cu::csrF & m)
        : handle(handle), mat(m), x(m.cols), b(m.rows)
    {
        auto xDesc = x.getCuSparseDescriptor();
        auto bDesc = b.getCuSparseDescriptor();
        matDesc = mat.getCuSparseDescriptor();

        size_t spmvBufferSize = 0;
        auto rc = hipsparseSpMV_bufferSize(handle, op,
                                          &alpha, matDesc, xDesc,
                                          &beta, bDesc,
                                          hipDataType::HIP_R_32F, hipsparseSpMVAlg_t::HIPSPARSE_SPMV_ALG_DEFAULT,
                                          &spmvBufferSize);
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseSpMV_bufferSize failed: {}", hipsparseGetErrorName(rc)));
        }
        std::cout << "Workspace buffer size: " << spmvBufferSize << "\n";

        workspace = cu::vec<char>(spmvBufferSize);

        rc = hipsparseSpMV_preprocess(handle, op,
                                     &alpha, matDesc, xDesc, &beta, bDesc,
                                     hipDataType::HIP_R_32F, hipsparseSpMVAlg_t::HIPSPARSE_SPMV_ALG_DEFAULT,
                                     workspace.get());
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseSpMV_preprocess failed: {}", hipsparseGetErrorName(rc)));
        }
    }

    spmv::~spmv()
    {
        
    }

    void spmv::compute()
    {
        return compute(x.getCuSparseDescriptor(), b.getCuSparseDescriptor());
    }

    void spmv::compute(hipsparseDnVecDescr_t otherX, hipsparseDnVecDescr_t otherB)
    {
        auto rc = hipsparseSpMV(handle, op, &alpha,
                               matDesc, otherX, &beta, otherB,
                               hipDataType::HIP_R_32F, hipsparseSpMVAlg_t::HIPSPARSE_SPMV_ALG_DEFAULT,
                               workspace.get());
        if (rc != hipsparseStatus_t::HIPSPARSE_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::format("hipsparseSpMV failed: {}", hipsparseGetErrorName(rc)));
        }
    }

    void spmv::compute(cu::vec<float> & otherX, cu::vec<float> & otherB)
    {
        auto xd = otherX.getCuSparseDescriptor();
        auto bd = otherB.getCuSparseDescriptor();
        compute(xd, bd);
    }

} // namespace cu