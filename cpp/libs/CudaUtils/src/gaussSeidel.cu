#include "hip/hip_runtime.h"
#include <cu/gaussSeidel.h>

#include <algorithm>
#include <array>
#include <cassert>
#include <stdexcept>

#include <linalg/gaussSeidel.h>
#include <linalg/graphs.h>

#include <utils/stopwatch.h>

#include <cu/stopwatch.h>

namespace cu
{
    __global__ void reorderXbFwd(const float * srcX, const float * srcB,
                                 float * dstX, float * dstB,
                                 const int * coloring, const int n)
    {
        int i0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int stride = blockDim.x * gridDim.x;
        for (int i = i0; i < n; i += stride)
        {
            const int j = coloring[i];
            dstX[i] = srcX[j];
            dstB[i] = srcB[j];
        }
    }

    __global__ void reorderXbFwd2ch(const float * srcX, const float * srcB,
                                    float * dstX, float * dstB,
                                    const int * coloring, const int n)
    {
        int i0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int stride = blockDim.x * gridDim.x;
        for (int i = i0; i < n; i += stride)
        {
            const int j = coloring[i];
            dstX[i] = srcX[j];
            dstB[i] = srcB[j];
            dstX[i + n] = srcX[j + n];
            dstB[i + n] = srcB[j + n];
        }
    }

    __global__ void reorderXInv(const float * srcX, float * dstX,
                                const int * coloring, const int n)
    {
        int i0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int stride = blockDim.x * gridDim.x;
        for (int i = i0; i < n; i += stride)
        {
            const int j = coloring[i];
            dstX[j] = srcX[i];
        }
    }

    __global__ void reorderXInv2ch(const float * srcX, float * dstX,
                                   const int * coloring, const int n)
    {
        int i0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int stride = blockDim.x * gridDim.x;
        for (int i = i0; i < n; i += stride)
        {
            const int j = coloring[i];
            dstX[j] = srcX[i];
            dstX[j + n] = srcX[i + n];
        }
    }

    // Perform a step of the Gauss-Seidel algorithm on a partition of the system Mx = b
    // The partition covers rows [partitionStart, partitionEnd)
    // The matrix is in CSR format, described by values, column and rowStart
    // It has no diagonal entries - the multiplicative inverse of the original matrix are stored in invDiag instead
    __global__ void gaussSeidelStepPartitionInvDiagR(float * x, const float * b, const float * invDiag,
                                                     const float * values, const int * column, const int * rowStart,
                                                     const int partitionStart, const int partitionEnd)
    {
        int row0 = partitionStart + blockIdx.x * blockDim.x + threadIdx.x;
        const int stride = blockDim.x * gridDim.x;
        for (int row = row0; row < partitionEnd; row += stride)
        {
            const int j1 = rowStart[row + 1];
            float negSum = 0;
            for (int j = rowStart[row]; j < j1; j++)
            {
                const int col = column[j];
                // col is never equal to row
                negSum += values[j] * x[col];
            }
            x[row] = (b[row] - negSum) * invDiag[row];
        }
    }

    __global__ void gaussSeidelStepPartitionInvDiagR2ch(float * x, const float * b, const float * invDiag,
                                                        const float * values, const int * column, const int * rowStart,
                                                        const int partitionStart, const int partitionEnd,
                                                        const int n)
    {
        int row0 = partitionStart + blockIdx.x * blockDim.x + threadIdx.x;
        const int stride = blockDim.x * gridDim.x;
        for (int row = row0; row < partitionEnd; row += stride)
        {
            const int j1 = rowStart[row + 1];
            float negSum0 = 0;
            float negSum1 = 0;
            for (int j = rowStart[row]; j < j1; j++)
            {
                const int col = column[j];
                // col is never equal to row
                negSum0 += values[j] * x[col];
                negSum1 += values[j] * x[col + n];
            }
            x[row] = (b[row] - negSum0) * invDiag[row];
            const int secondRow = row + n;
            x[secondRow] = (b[secondRow] - negSum1) * invDiag[row];
        }
    }

    GaussSeidel::GaussSeidel(cu::Blas & blas, hipsparseHandle_t sparseHandle, const linalg::CsrMatrix<float> & cpuMatrix,
                             const int numCh)
        : blas(blas),
          coloring(cpuMatrix.cols),
          rhs(cpuMatrix.cols * numCh), sol(cpuMatrix.cols * numCh),
          numCh(numCh),
          ioRhs(cpuMatrix.cols * numCh), ioSol(cpuMatrix.cols * numCh)
    {
        assert(cpuMatrix.cols == cpuMatrix.rows);
        const int n = cpuMatrix.cols;

        if (numCh < 1 || numCh > 2)
        {
            throw std::invalid_argument("Only 1 and 2 ch Gauss-Seidel is supported");
        }

        // TODO Extract the matrix preprocessing to separate function?
        // Create a coloring of the matrix
        // Use the smallest-last ordering for now - it seems to produce good results
        auto graph = linalg::buildCsrGraph(cpuMatrix);
        assert(graph.size() == n);
        auto slOrder = linalg::buildSmallestLastOrdering(graph);
        auto parts = linalg::partitionGraphGreedy(graph, slOrder);

        const int nColors = parts.size();

        // Sort the individual partitions and place them in the coloring vector
        cpuColoring.resize(n);
        partitionStart.resize(nColors + 1);
        partitionStart.back() = n;
        int i = 0;
        for (int c = 0; c < nColors; c++)
        {
            auto & p = parts[c];
            std::sort(p.begin(), p.end());
            std::copy_n(p.begin(), p.size(), cpuColoring.begin() + i);
            partitionStart[c] = i;
            i += p.size();
        }

        // Reorder the matrix to make the coloring redundant -
        // first partition is [0, P1), second is [P1, P2) and so on
        auto reordered = cpuMatrix.slice(cpuColoring, cpuColoring);

        // Upload the reordered matrix
        m = std::make_unique<csrF>(reordered);

        if (numCh == 1)
        {
            mSpmv = std::make_unique<spmv>(sparseHandle, *m);
        }
        else
        {
            mSpmm = std::make_unique<spmm>(sparseHandle, *m, numCh);
        }

        // Create a stripped matrix (no diagonal) and the inverted diagonal
        auto ctx = linalg::buildGaussSeidelContext(reordered);
        invDiag.overwriteUpload(ctx.invDiag);
        values.overwriteUpload(ctx.stripped.values);
        column.overwriteUpload(ctx.stripped.column);
        rowStart.overwriteUpload(ctx.stripped.rowStart);

        // Upload the coloring
        coloring.upload(cpuColoring);

        // Calculate kernel sizes
        constexpr int maxThreads = 512;
        if (n <= maxThreads)
        {
            reorderBlockSize = dim3(n);
            reorderGridSize = dim3(1);
        }
        else
        {
            const int nB = (n + maxThreads - 1) / maxThreads;
            reorderBlockSize = dim3(maxThreads);
            reorderGridSize = dim3(nB);
        }

        // Calculate block and grid sizes for each partition
        const int nParts = partitionStart.size() - 1;
        blockSize.resize(nParts);
        gridSize.resize(nParts);
        for (int p = 0; p < nParts; p++)
        {
            const int pSize = partitionStart[p + 1] - partitionStart[p];
            if (pSize <= maxThreads)
            {
                blockSize[p] = dim3(pSize);
                gridSize[p] = dim3(1);
            }
            else
            {
                blockSize[p] = dim3(maxThreads);
                const int nBlocks = (pSize + maxThreads - 1) / maxThreads;
                gridSize[p] = dim3(nBlocks);
            }
        }

        lastMse.resize(numCh);
    }

    float GaussSeidel::solve1(const int maxIters, const float target)
    {
        const int n = coloring.size();
        const int nParts = partitionStart.size() - 1;

        // Reorder the IO vectors
        reorderXbFwd<<<reorderGridSize, reorderBlockSize>>>(ioSol.get(), ioRhs.get(),
                                                            sol.get(), rhs.get(),
                                                            coloring.get(), n);

        float lastMse = -1;

#ifdef CU_SOLVERS_ITER_LOG
        Stopwatch sw;
        u::Stopwatch bigSw;
#endif

        int iter = 0;
        for (; iter < maxIters; iter++)
        {
#ifdef CU_SOLVERS_ITER_LOG
            bigSw.reset();
            sw.reset();
#endif

            // Perform the updates
            for (int p = 0; p < nParts; p++)
            {
                const int j0 = partitionStart[p];
                const int j1 = partitionStart[p + 1];

                // Send it
                const dim3 currGrid = gridSize[p];
                const dim3 currBlock = blockSize[p];
                gaussSeidelStepPartitionInvDiagR<<<currGrid, currBlock>>>(sol.get(), rhs.get(), invDiag.get(),
                                                                          values.get(), column.get(), rowStart.get(),
                                                                          j0, j1);
            }

#ifdef CU_SOLVERS_ITER_LOG
            const auto tGs = sw.millis(true);
#endif

            bool done = false;
            float mse = lastMse;
            if (iter % mseMod == 0)
            {
                // Calculate MSE
                auto & res = mSpmv->b;
                mSpmv->compute(sol, res);
                const int n = sol.size();
                cu::saxpy(blas, n, rhs.get(), res.get(), -1.0f);

                float norm2 = -1; // == sqrt(sum(res[i]^2))
                auto rc = hipblasSnrm2(blas.handle, n, res.get(), 1, &norm2);
                if (rc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
                {
                    throw std::runtime_error(std::format("hipblasSnrm2 failed: {}", cublasGetStatusName(rc)));
                }

                mse = norm2 / std::sqrt(n);
                lastMse = mse;
                if (mse < target)
                {
                    done = true;
                }
            }

#ifdef CU_SOLVERS_ITER_LOG
            const auto tMse = sw.millis();
            const auto tIter = bigSw.millis();
            std::cout << iter << ": " << mse << " (gs = " << tGs << " ms, mse = " << tMse << " ms, total = " << tIter << " ms)\n";
#endif
            if (done)
            {
                break;
            }
        }

        // Place the result in the IO vector
        reorderXInv<<<reorderGridSize, reorderBlockSize>>>(sol.get(), ioSol.get(), coloring.get(), n);

        this->lastMse[0] = lastMse;
        this->lastIterations = iter;

        return lastMse;
    }

    float GaussSeidel::solve2(const int maxIters, const float target)
    {
        const int n = coloring.size();
        const int nParts = partitionStart.size() - 1;

        // Reorder the IO vectors
        reorderXbFwd2ch<<<reorderGridSize, reorderBlockSize>>>(ioSol.get(), ioRhs.get(),
                                                               sol.get(), rhs.get(),
                                                               coloring.get(), n);
        std::array<float, 2> lastMse = {-1, -1};

#ifdef CU_SOLVERS_ITER_LOG
        Stopwatch sw;
        u::Stopwatch bigSw;
#endif

        int iter = 0;
        for (; iter < maxIters; iter++)
        {
#ifdef CU_SOLVERS_ITER_LOG
            bigSw.reset();
            sw.reset();
#endif

            // Perform the updates
            for (int p = 0; p < nParts; p++)
            {
                const int j0 = partitionStart[p];
                const int j1 = partitionStart[p + 1];

                // Send it
                const dim3 currGrid = gridSize[p];
                const dim3 currBlock = blockSize[p];
                gaussSeidelStepPartitionInvDiagR2ch<<<currGrid, currBlock>>>(sol.get(), rhs.get(), invDiag.get(),
                                                                             values.get(), column.get(), rowStart.get(),
                                                                             j0, j1, n);
            }

#ifdef CU_SOLVERS_ITER_LOG
            const auto tGs = sw.millis(true);
#endif

            bool done = false;
            std::array<float, 2> mse = {lastMse[0], lastMse[1]};
            if (iter % mseMod == 0)
            {
                // Calculate MSE
                auto & resXy = mSpmm->b;
                mSpmm->compute(sol, resXy);
                cu::saxpy(blas, 2 * n, rhs.get(), resXy.get(), -1.0f);

                for (int c = 0; c < 2; c++)
                {
                    float norm2 = -1; // == sqrt(sum(res[i]^2))
                    auto rc = hipblasSnrm2(blas.handle, n, resXy.get() + c * n, 1, &norm2);
                    if (rc != hipblasStatus_t::HIPBLAS_STATUS_SUCCESS)
                    {
                        throw std::runtime_error(std::format("hipblasSnrm2 failed: {}", cublasGetStatusName(rc)));
                    }

                    mse[c] = norm2 / std::sqrt(n);
                    lastMse[c] = mse[c];
                }

                if (mse[0] < target && mse[1] < target)
                {
                    done = true;
                }
            }

#ifdef CU_SOLVERS_ITER_LOG
            const auto tMse = sw.millis();
            const auto tIter = bigSw.millis();
            std::cout << iter << ": " << mse[0] << " / " << mse[1] << " (gs = " << tGs << " ms, mse = " << tMse << " ms, total = " << tIter << " ms)\n";
#endif
            if (done)
            {
                break;
            }
        }

        // Place the result in the IO vector
        reorderXInv2ch<<<reorderGridSize, reorderBlockSize>>>(sol.get(), ioSol.get(), coloring.get(), n);

        // Average the channel MSE
        const float avgMse = std::sqrt(lastMse[0] * lastMse[0] + lastMse[1] * lastMse[1]);

        this->lastMse[0] = lastMse[0];
        this->lastMse[1] = lastMse[1];
        this->lastIterations = iter;

        return avgMse;
    }

    void GaussSeidel::setMseCheckInterval(const int newInterval)
    {
        if (newInterval < 1)
        {
            throw std::invalid_argument("MSE check interval should be at least 1");
        }

        mseMod = newInterval;
    }

    float GaussSeidel::solve(const int maxIters, const float target)
    {
        if (numCh == 1)
        {
            return solve1(maxIters, target);
        }
        else
        {
            assert(numCh == 2);
            return solve2(maxIters, target);
        }
    }
} // namespace cu
