#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <vector>

#include <cu/vec.h>
#include <cu/csr.h>

__global__ void mul10(int * vec, const int size)
{
    int i0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = i0; i < size; i += stride)
    {
        vec[i] *= vec[i];
    }
}

void test()
{
    const int n = 100;
    std::vector<int> cpuVec(n);
    std::default_random_engine rng(std::random_device{}());
    std::uniform_int_distribution<int> dist(0, 20);
    for (int i = 0; i < n; i++)
    {
        cpuVec[i] = dist(rng);
        std::cout << cpuVec[i] << " ";
    }
    std::cout << "\n";

    cu::vec<int> gpuVec(n);
    gpuVec.uploadAsync(cpuVec);
    // auto rc = hipMemcpy(gpuVec.get(), cpuVec.data(), n * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);
    // if (rc != hipSuccess)
    // {
    //     std::cerr << "Failed to memcpy: " << rc << "\n";
    //     return;
    // }

    const dim3 blockSize(1);
    const dim3 gridSize(1);
    mul10<<<gridSize, blockSize>>>(gpuVec.get(), gpuVec.size());

    // rc = hipMemcpy(cpuVec.data(), gpuVec.get(), n * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
    // if (rc != hipSuccess)
    // {
    //     std::cerr << "Failed to memcpy: " << rc << "\n";
    //     return;
    // }
    gpuVec.downloadAsync(cpuVec);

    auto rc = hipStreamSynchronize(0);
    if (rc != hipSuccess)
    {
        std::cerr << "Sync failed: " << rc << "\n";
        return;
    }

    for (int i = 0; i < n; i++)
    {
        std::cout << cpuVec[i] << " ";
    }
    std::cout << "\n";
}